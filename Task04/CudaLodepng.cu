#include "hip/hip_runtime.h"
//Muhammad Iqbal
//1718170

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include "lodepng.h"

//each pixel has 4 values:
//Red (R)
//Green (G)
//Blue (B)
//Transparency (T)

//negative filter (reversing the RGB values)
//0-255   max-val    255-40

struct RGBT {
	int r;
	int g;
	int b;
	int t;
};

const int MAX_VALUE = 255;
const int ERROR_EXIT_VALUE = -1;


void printRGBTArray(unsigned char* rgbtArray, int height, int width)
{
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width*4; col += 4) {
			printf("Row: '%d' Col: '%d' R:%d, G:%d, B:%d, A:%d\n", row, col / 4, rgbtArray[row*width*4+col], rgbtArray[row*width*4+col+1], rgbtArray[row*width*4+col+2], rgbtArray[row*width*4+col+3]);
		}
	}
	printf("\n");
}

__device__
struct RGBT GetAverage(struct RGBT* vals, int valsLength, int totalValidValues)
{
	double totalR = 0.0, totalG = 0.0, totalB = 0.0;
	for (int i = 0; i < valsLength; i++) {
		totalR += vals[i].r;
		totalG += vals[i].g;
		totalB += vals[i].b;
	}

	struct RGBT getAverageValues;
	getAverageValues.r = totalR / totalValidValues;
	getAverageValues.g = totalG / totalValidValues;
	getAverageValues.b = totalB / totalValidValues;
	getAverageValues.t = MAX_VALUE;
	return getAverageValues;
}

__device__
struct RGBT GetRGBTValuesAtPixel(unsigned char* inputImage, int pixel) {
	struct RGBT values;
	values.r = inputImage[pixel];
	values.g = inputImage[pixel+1];
	values.b = inputImage[pixel+2];
	values.t = inputImage[pixel+3];
	return values;
}


__global__ 
void NegativeFilter(unsigned char* actualVals, unsigned char* blurredChars, int width, int height) {
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

	int pixelY = threadIndex % width;
	int pixelX = threadIndex / width;

	struct RGBT* thisValues = (struct RGBT*) malloc( sizeof(struct RGBT) * 9);
	int arrayStartIndex = threadIndex * 4;

	int validValuesCount = 0;
	thisValues[4] = GetRGBTValuesAtPixel(actualVals, arrayStartIndex);
	validValuesCount++;

	bool atLeftWall = pixelY == 0;
	bool atTopWall = pixelX == 0;
	bool atRightWall = pixelY > 0 && pixelY % (width - 1) * 4 == 0;
	bool atBtmWall = pixelX >= height - 1;

	int colArrayIndex = pixelY * 4;

	if (!atLeftWall) {
		int mlIndex = (pixelX * width * 4) + (colArrayIndex - (4 * 1));
		thisValues[3] = GetRGBTValuesAtPixel(actualVals, mlIndex);
		validValuesCount++;
	
	
	if (!atTopWall) {
		int tlIndex = ((pixelX + 1) * width * 4) + (colArrayIndex - (4 * 1));
		thisValues[6] = GetRGBTValuesAtPixel(actualVals, tlIndex);
		validValuesCount++;
	}

	if (!atBtmWall) {
		int blIndex = ((pixelX + 1) * width * 4) + (colArrayIndex - (4 * 1));
		thisValues[6] = GetRGBTValuesAtPixel(actualVals, blIndex);
		validValuesCount++;
	}
}




	if (!atRightWall) {
		int mrIndex = (pixelX * width * 4) + (colArrayIndex + (4 * 1));
		thisValues[5] = GetRGBTValuesAtPixel(actualVals, mrIndex);
		validValuesCount++;

	if (!atTopWall) {
		int trIndex = ((pixelX - 1 ) * width * 4) + (colArrayIndex + (4 * 1));
		thisValues[2] = GetRGBTValuesAtPixel(actualVals, trIndex);
		validValuesCount++;
	}

	if (!atBtmWall) {
		int brIndex = ((pixelX + 1 ) * width * 4) + (colArrayIndex + (4 * 1));
		thisValues[8] = GetRGBTValuesAtPixel(actualVals, brIndex);
		validValuesCount++;
	}
}



	if (!atTopWall) {
		int tmIndex = ((pixelX - 1 ) * width * 4) + colArrayIndex;
		thisValues[1] = GetRGBTValuesAtPixel(actualVals, tmIndex);
		validValuesCount++;
	}

	if (!atBtmWall) {
		int bmIndex = ((pixelX + 1 ) * width * 4) + colArrayIndex;
		thisValues[7] = GetRGBTValuesAtPixel(actualVals, bmIndex);
		validValuesCount++;
	}

	struct RGBT blurredVals = GetAverage(thisValues, 9, validValuesCount);
	blurredChars[arrayStartIndex] = blurredVals.r;
	blurredChars[arrayStartIndex + 1] = blurredVals.g;
	blurredChars[arrayStartIndex + 2] = blurredVals.b;
	blurredChars[arrayStartIndex + 3] = blurredVals.t;
	free(thisValues);
}

/**
printf("uid = %d\n", pixel);

r = inputImage[pixel];
g = inputImage[pixel+1];
b = inputImage[pixel+2];
t = inputImage[pixel+3];

outputImage[pixel] = 255-r;
outputImage[pixel+1] = 255-g;
outputImage[pixel+2] = 255-b;
outputImage[pixel+3] = t;

}**/

int main(int argc, char ** argv){

  unsigned int errorDecode; //variable will hold whether there was an issue with loading in the png file
  unsigned char* cpuImage; /** = (unsigned char*) malloc( sizeof(unsigned char) * width * height * 4 errorDecode = lodepng_decode32_file(&cpuImage, &width, &height, filename); **/ //this variable will hold all of our image data
  unsigned int width, height; //holds the width and height of image
  
  char* filename = "1.png";
  if (argc>1)
  filename = argv[1];

  char* newFilename = "blurredimg.png";
  if (argc>2)
  newFilename = argv[2];

  errorDecode = lodepng_decode32_file(&cpuImage, &width, &height, filename); // (where to store the image data, width, height, which file?)
  if(errorDecode){
    printf("error %u: %s\n", errorDecode, lodepng_error_text(errorDecode));
    exit(ERROR_EXIT_VALUE);
  }
  
  int arraySize = width*height*4;
  if (width <=0 || height <= 0) {
  printf("This is unable to decode the image. Please double check the png file and try again\n");
  exit(ERROR_EXIT_VALUE);
}
  //int memorySize = arraySize * sizeof(unsigned char);
  
  //unsigned char cpuOutImage[arraySize];
  
  unsigned char* gpuInput;
  hipMalloc( (void**) &gpuInput, sizeof(unsigned char) * arraySize);
  hipMemcpy(gpuInput, cpuImage, sizeof(unsigned char) * arraySize, hipMemcpyHostToDevice);

  unsigned char* gpuOutput;
  hipMalloc( (void**) &gpuOutput, sizeof(unsigned char) * arraySize);
  
  //hipMemcpy(gpuInput, cpuImage, memorySize, hipMemcpyHostToDevice);
  
  NegativeFilter<<< dim3(width,1,1), dim3(height,1,1) >>>(gpuInput, gpuOutput, width, height);
  hipDeviceSynchronize();
  unsigned char* cpuOutImage = (unsigned char*) malloc(sizeof(unsigned char) * arraySize);
  hipMemcpy(cpuOutImage, gpuOutput, sizeof(unsigned char) * arraySize, hipMemcpyDeviceToHost);
  
  unsigned int errorEncode = lodepng_encode32_file(newFilename, cpuOutImage, width, height);
  if(errorEncode) {
  printf("error %u: %s\n", errorEncode, lodepng_error_text(errorEncode));
  exit(ERROR_EXIT_VALUE);
  }

  //free(image);
  free(cpuImage);
  free(cpuOutImage);
  hipFree(gpuInput);
  hipFree(gpuOutput);

}


