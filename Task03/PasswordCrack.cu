//Muhammad Iqbal
//1718170


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables

__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

/**
*	Checking to see if first string matches with other length char strings 
*/
__device__
int MatchingEcryption(char* first, char* second, int length) {
	int result = 1;
	for (int i = 0; i < length; i++) {
		if (first[i] != second[i]) {
			result = 0;
			break;
		}
	}
	return result;
}

/**
*	The pass to be Decrypted via a coda thread
*/
__global__
void crack(char* alphabet, char* numbers, char* encryptedPsw, char* passOutput){
	int uid = blockDim.x * blockIdx.x + threadIdx.x;

	if(*passOutput != NULL) {
		return;
	}

	char genRawPass[4];

	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];

	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];

	//firstLetter - 'a' - 'z' (26 characters)
	//secondLetter - 'a' - 'z' (26 characters)
	//firstNum - '0' - '9' (10 characters)
	//secondNum - '0' - '9' (10 characters)

//Idx --> gives current index of the block or thread
	
	char* potentialEncry;
	potentialEncry = CudaCrypt(genRawPass);

	if (MatchingEcryption(encryptedPsw, potentialEncry, 11) > 0 )
	{
		
		for (int i = 0; i < 4; i++) {
		passOutput[i] = genRawPass[i];
		}
	}

}

int main(int argc, char ** argv){

	char * encryptedPsw = "ccbddb7362";
	if (argc > 1) {
		encryptedPsw = argv[1];
	}
	printf("Password Given: '%s'\n", encryptedPsw);

	char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
	char cpuNumbers[10] = {'0','1','2','3','4','5','6','7','8','9'};
	int encryptedPswSize = sizeof(char) * 11;

	char * gpuAlphabet;
	hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
	hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);



	char * gpuNumbers;
	hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10); 
	hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);


	char * gpuOutputPass;
	hipMalloc( (void**) &gpuOutputPass, encryptedPswSize);


	char * gpuencryptedPsw;
	hipMalloc( (void**) &gpuencryptedPsw, encryptedPswSize);
	hipMemcpy( gpuencryptedPsw, encryptedPsw, encryptedPswSize, hipMemcpyHostToDevice);


	crack<<< dim3(26, 26, 1), dim3(10, 10, 1) >>>( gpuAlphabet, gpuNumbers, gpuencryptedPsw, gpuOutputPass);
	hipDeviceSynchronize();


	char* cpuPassOutput = (char*)malloc( sizeof(char) * 4);
	hipMemcpy(cpuPassOutput, gpuOutputPass, encryptedPswSize, hipMemcpyDeviceToHost);
	if (cpuPassOutput != NULL && cpuPassOutput[0] != 0) {
		printf("Given Encrypted Password: '%s'\n", encryptedPsw);
		printf("Found Decrypted Password: '%s'\n", cpuPassOutput);
	
	} else {
		printf("Password could not be determined.\n");
	}

	hipFree(gpuAlphabet);
	hipFree(gpuNumbers);
	hipFree(gpuencryptedPsw);
	hipFree(gpuOutputPass);
	free(cpuPassOutput);
}









